#include "hip/hip_runtime.h"
/**
 * Name: Kim Seonggyeom 
 * Student id: 20668626
 * ITSC email: skimbc@connect.ust.hk
*/

#include <iostream>
#include <fstream>
#include <string>
#include <algorithm>
#include <chrono>
#include <cassert>
#include <stdio.h>

#include "cuda_smith_waterman.h"

using namespace std;

__global__ void init_score_matrix(int *d_a_len, int *d_b_len, int *d_score)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int numThreads = gridDim.x * blockDim.x;

    int atdia_len = min(*d_a_len, *d_b_len) + 2;
    int total_len = 3 * atdia_len;

    for (int i = threadID; i < total_len; i += numThreads)
    {
        d_score[i] = 0;
    }
}
__global__ void init_max_scores(int *d_max_scores){
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    d_max_scores[threadID] = 0;
}
__global__ void smith_waterman_per_row(int first_atdia_idx, int second_atdia_idx, int current_atdia_idx, int cnt, int i, int j, char *d_a, char *d_b, int *d_score, int *d_max_scores)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int numThreads = gridDim.x * blockDim.x;

    for (int l = threadID + 1; l <= cnt; l += numThreads)
    {
        d_score[current_atdia_idx + l] = max(0,
                                             max(d_score[first_atdia_idx + l - 1] + sub_mat(d_a[i - (l - 1) - 1], d_b[j + (l - 1) - 1]),
                                                 max(d_score[second_atdia_idx + l - 1] - GAP,
                                                     d_score[second_atdia_idx + l] - GAP)));

        d_max_scores[threadID] = max(d_max_scores[threadID], d_score[current_atdia_idx + l]);
    }
}
int smith_waterman(int blocks_per_grid, int threads_per_block, char *a, char *b, int a_len, int b_len)
{
    int max_score = 0;
    int *max_scores = (int *)malloc(sizeof(int) * (threads_per_block * blocks_per_grid));

    dim3 blocks(blocks_per_grid);
    dim3 threads(threads_per_block);

    int num_iter = a_len + b_len - 1;
    int atdia_len = min(a_len, b_len) + 2; // anti diagonal length

    int first_atdia_idx = 0; // first anti diagonal index
    int second_atdia_idx = atdia_len; // second anti diagonal index

    char *d_a;
    char *d_b;
    int *d_a_len;
    int *d_b_len;
    int *d_score;
    int *d_max_scores; // local max scores per thread

    hipMalloc(&d_a, sizeof(char) * (a_len + 1));
    hipMalloc(&d_b, sizeof(char) * (b_len + 1));
    hipMalloc(&d_a_len, sizeof(int));
    hipMalloc(&d_b_len, sizeof(int));
    hipMalloc(&d_score, sizeof(int) * (3 * atdia_len));
    hipMalloc(&d_max_scores, sizeof(int) * (threads_per_block * blocks_per_grid));

    hipMemcpy(d_a, a, sizeof(char) * (a_len + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(char) * (b_len + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_a_len, &a_len, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_len, &b_len, sizeof(int), hipMemcpyHostToDevice);

    init_score_matrix<<<blocks, threads>>>(d_a_len, d_b_len, d_score);
    hipDeviceSynchronize();

    init_max_scores<<<blocks, threads>>>(d_max_scores);
    hipDeviceSynchronize();

    // travese each anti diagonal row from top left to bottom right
    for (int k = 1; k <= num_iter; k++)
    {
        int i = min(k, a_len);
        int j = max(0, k - a_len) + 1;
        int cnt = min(k, min(b_len - j + 1, i));
        if (j > 1)
        {
            first_atdia_idx++;
            second_atdia_idx++;
        }
        int current_atdia_idx = ((k+1) % 3) * atdia_len;
        
        // process each anti diagonal row using GPU
        smith_waterman_per_row<<<blocks, threads>>>(first_atdia_idx, second_atdia_idx, current_atdia_idx, cnt, i, j, d_a, d_b, d_score, d_max_scores);
        hipDeviceSynchronize();

        // change index
        first_atdia_idx = second_atdia_idx; 
        second_atdia_idx = current_atdia_idx; 
    }
    hipMemcpy(max_scores, d_max_scores, sizeof(int) * (threads_per_block * blocks_per_grid), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    for (int i = 0; i < threads_per_block * blocks_per_grid; i++)
    {
        max_score = max(max_score, max_scores[i]);
    }
    free(max_scores);

    return max_score;
}